
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5
#define M 10

//global means it is called by host, run by device
//mat is the original matrix *already allocated on GPU*
//mat_res is the matrix to store the result *already allocated on GPU*
//s is the scalar, passed directly from host to function
__global__ 
void mat_mult(int *mat, int *mat_res, int *mult)
{
	//row
	int tidX = blockIdx.x * blockDim.x + threadIdx.x;

	//col
	int tidY = blockIdx.y * blockDim.y + threadIdx.y;

	//thread ID must be < # of matrix rows and columns
	if(tidX < M && tidY < N)
		mat_res[tidX * N  + tidY] = mat[tidX * N + tidY] * mult[tidY];
}

//__host__ is default (called and run on host), so this is optional
__host__
int main()
{
	//host stuff
	int *mat = (int *) malloc(N * M * sizeof(int));
	int *mat_res = (int *) malloc(N * M * sizeof(int));

	int *mult = (int *) malloc(N * sizeof(int));
	int *mult_res = (int *) malloc(M * sizeof(int));


	//device stuff
	int *d_mat, *d_mat_res, *d_mult, *d_mult_res;

	printf("Past Pointer Var Dec\n");

	//fill host matrix
	int i, j;
	for(i = 0; i < M; i++)
		for(j = 0; j < N; j++)
			mat[i * M + j] = i * N + j;

	for(i = 0; i < N; i++)
		mult[i] = 20 + i;

	printf("Original matrix...\n");
	for(i = 0; i < M; i++)
	{
		for(j = 0; j < N; j++)
			printf("%d\t", mat[i * M + j]);
		printf("\n");
	}
	
	printf("Allocating CUDA memory\n");
	//allocate device memory
	hipMalloc((void **) &d_mat,N * M * sizeof(int));
	hipMalloc((void **) &d_mat_res, N * M * sizeof(int));

	printf("1\n");

	hipMalloc((void **) &d_mult, N * sizeof(int));
	hipMalloc((void **) &d_mult_res, M * sizeof(int));
	

	//copy host matrix to device
	printf("Copying to device...\n");
	hipMemcpy(d_mat, mat, N * M * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_mult, mult, N * sizeof(int), hipMemcpyHostToDevice);

	printf("Starting kernel...\n");
	
	//specify the number of threads per block in X and Y dimensions
	dim3 dimBlock(16, 16, 1);
		
	//specify the number of blocks: we need enough blocks in both the X and Y
	// dimensions to cover the entire matrix, assuming we have 16 threads/block
	dim3 dimGrid((M - 1)/16 + 1, (N - 1)/16 + 1, 1);

	//call the kernel
	mat_mult<<<dimGrid, dimBlock>>>(d_mat, d_mat_res, d_mult);

	printf("Copying back...\n");

	hipMemcpy(mat_res, d_mat_res, N * M * sizeof(int), hipMemcpyDeviceToHost);

	printf("Final matrix...\n");
	for(i = 0; i < M; i++)
	{
		for(j = 0; j < N; j++)
			printf("%d\t", mat_res[i * M + j]);
		printf("\n");
	}
	
	return 0;
}
